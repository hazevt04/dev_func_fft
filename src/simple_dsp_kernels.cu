#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_complex.h>
#include <hip/hip_cooperative_groups.h>

#include "simple_dsp_kernels.cuh"


// FFT Implementation from C++ Cookbook:
// https://www.oreilly.com/library/view/c-cookbook/0596007612/ch11s18.html#cplusplusckbk-CHP-11-EX-33
__device__ 
unsigned int bit_reverse(unsigned int x, int log2n) {
   unsigned int n = 0;
   for (int i = 0; i < log2n; i++) {
      n <<= 1;
      n |= (x & 1);
      x >>= 1;
   }
   return n;
}


__device__ 
float complex_phase_angle(const hipfftComplex& val) { 
   return atan2( hipCimagf(val), hipCrealf(val)); 
} 


__global__
void calc_con_sqrs(hipfftComplex* __restrict__ con_sqrs, const hipfftComplex* __restrict__ frequencies, const int num_frequencies) {

   //Assuming one stream
   int global_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   
   for (int index = global_index; index < num_frequencies; index+=stride) {   
      hipfftComplex conj = hipConjf(frequencies[index]);
      con_sqrs[index] = hipCmulf( conj, conj );
   }
}


__global__ 
void calc_psds(float* __restrict__ psds, const hipfftComplex* __restrict__ con_sqrs, const int num_con_sqrs, const float log10num_con_sqrs) {
   
   // Assuming one stream
   int global_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   
   for (int index = global_index; index < num_con_sqrs; index+=stride) {   
      psds[index] = 10*__log10f( hipCabsf(con_sqrs[index]) ) - log10num_con_sqrs;
      
   }

}

namespace cg = cooperative_groups;  

__global__
void cookbook_fft64(hipfftComplex* frequencies, const hipfftComplex* __restrict__ samples, const int num_samples) {
   auto group = cg::this_thread_block();

   for (int index = group.thread_rank(); index < num_samples; index += group.size() ) {
      const hipfftComplex J = make_hipComplex(0,-1);

      int br_index = (int)bit_reverse((int)index, NUM_FFT_SIZE_BITS);
      frequencies[index].x = samples[br_index].x;
      frequencies[index].y = samples[br_index].y;

      for (int s = 1; s <= NUM_FFT_SIZE_BITS; ++s) {
         unsigned int m = (1 << s);
         unsigned int m2 = (m >> 1);
         hipfftComplex w = make_hipComplex(1, 0);
         hipfftComplex wm = complex_exponential( hipCmulf( J, make_hipComplex( (PI / m2), 0 ) ) );
         for (unsigned int j = 0; j != m2; ++j) {
            for (int k = j; k < FFT_SIZE; k += m) {
               hipfftComplex t = hipCmulf( w, frequencies[k + m2] );
               hipfftComplex u = make_hipComplex( frequencies[k].x, frequencies[k].y );
               group.sync();
               frequencies[k] = hipCaddf( u, t );
               frequencies[k + m2] = hipCsubf( u, t );
               group.sync();
            }
            w = hipCmulf( w, wm );
         } // end of for (unsigned int j = 0; j != m2; ++j) {
      } // end of for (int s = 1; s <= NUM_FFT_SIZE_BITS; ++s) {
   } // end of for (int index = grid.thread_rank(); index < num_samples; index += grid.size() ) {
      
} // end of cookbook_fft64


__global__
void simple_dsp_kernel(float* __restrict__ psds, hipfftComplex* __restrict__ con_sqrs, hipfftComplex* frequencies, 
      const hipfftComplex* __restrict__ samples, const int num_samples, const float log10num_con_sqrs) {
  
   auto group = cg::this_thread_block();
   if ( group.thread_rank() == 0 ) {
      cookbook_fft64<<<1, group.size()>>>( frequencies, samples, num_samples );
   }
}

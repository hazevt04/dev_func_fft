#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_complex.h>
#include <hip/hip_cooperative_groups.h>

#include "simple_dsp_kernels.cuh"


__device__ 
float complex_phase_angle(const hipfftComplex& val) { 
   return atan2( hipCimagf(val), hipCrealf(val)); 
} 

namespace cg = cooperative_groups;  

__device__
void calc_con_sqrs(hipfftComplex* __restrict__ con_sqrs, const hipfftComplex* __restrict__ frequencies, const int num_frequencies) {

   auto group = cg::this_thread_block();
   
   for (int index = group.thread_rank(); index < num_frequencies; index += group.size() ) {
      hipfftComplex conj = hipConjf(frequencies[index]);
      con_sqrs[index] = hipCmulf( conj, conj );
   }
}


__device__ 
void calc_psds(float* __restrict__ psds, const hipfftComplex* __restrict__ con_sqrs, const int num_con_sqrs, const float log10num_con_sqrs) {
   
   auto group = cg::this_thread_block();
   
   for (int index = group.thread_rank(); index < num_con_sqrs; index += group.size() ) {
      psds[index] = 10*__log10f( hipCabsf(con_sqrs[index]) ) - log10num_con_sqrs;
   }

}



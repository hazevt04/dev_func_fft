#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_complex.h>
#include <hip/hip_cooperative_groups.h>

#include "simple_dsp_kernels.cuh"


// FFT Implementation from C++ Cookbook:
// https://www.oreilly.com/library/view/c-cookbook/0596007612/ch11s18.html#cplusplusckbk-CHP-11-EX-33
__device__ 
unsigned int bit_reverse(unsigned int x, int log2n) {
   unsigned int n = 0;
   for (int i = 0; i < log2n; i++) {
      n <<= 1;
      n |= (x & 1);
      x >>= 1;
   }
   return n;
}


__device__ 
float complex_phase_angle(const hipfftComplex& val) { 
   return atan2( hipCimagf(val), hipCrealf(val)); 
} 


__global__
void calc_con_sqrs(hipfftComplex* __restrict__ con_sqrs, const hipfftComplex* __restrict__ frequencies, const int num_frequencies) {

   //Assuming one stream
   int global_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   
   for (int index = global_index; index < num_frequencies; index+=stride) {   
      hipfftComplex conj = hipConjf(frequencies[index]);
      con_sqrs[index] = hipCmulf( conj, conj );
   }
}


__global__ 
void calc_psds(float* __restrict__ psds, const hipfftComplex* __restrict__ con_sqrs, const int num_con_sqrs, const float log10num_con_sqrs) {
   
   // Assuming one stream
   int global_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   
   for (int index = global_index; index < num_con_sqrs; index+=stride) {   
      psds[index] = 10*__log10f( hipCabsf(con_sqrs[index]) ) - log10num_con_sqrs;
      
   }

}

namespace cg = cooperative_groups;  

__device__
void cookbook_fft64(hipfftComplex* frequencies, hipfftComplex* __restrict__ sh_samples, const int num_samples) {
   auto group = cg::this_thread_block();

   for (int index = group.thread_rank(); index < num_samples; index += group.size() ) {

      int br_index = (int)bit_reverse((int)index, NUM_FFT_SIZE_BITS);
      sh_samples[index].x = sh_samples[br_index].x;
      sh_samples[index].y = sh_samples[br_index].y;

      const hipfftComplex J = make_hipComplex(0,-1);
      for (int s = 1; s <= NUM_FFT_SIZE_BITS; ++s) {
         unsigned int m = (1 << s);
         unsigned int m2 = (m >> 1);
         hipfftComplex w = make_hipComplex(1, 0);
         hipfftComplex wm = complex_exponential( hipCmulf( J, make_hipComplex( (PI / m2), 0 ) ) );
         for (unsigned int j = 0; j != m2; ++j) {
            for (int k = j; k < FFT_SIZE; k += m) {
               hipfftComplex t = hipCmulf( w, sh_samples[k + m2] );
               hipfftComplex u = make_hipComplex( sh_samples[k].x, sh_samples[k].y );
               group.sync();
               sh_samples[k] = hipCaddf( u, t );
               sh_samples[k + m2] = hipCsubf( u, t );
               group.sync();
            }
            w = hipCmulf( w, wm );
         } // end of for (unsigned int j = 0; j != m2; ++j) {
      } // end of for (int s = 1; s <= NUM_FFT_SIZE_BITS; ++s) {
      frequencies[index].x = sh_samples[index].x;
      frequencies[index].y = sh_samples[index].y;
   } // end of for (int index = grid.thread_rank(); index < num_samples; index += grid.size() ) {
   
} // end of cookbook_fft64


__global__
void simple_dsp_kernel(float* __restrict__ psds, hipfftComplex* __restrict__ con_sqrs, hipfftComplex* frequencies, 
      const hipfftComplex* __restrict__ samples, const int num_samples, const float log10num_con_sqrs) {
  
   extern __shared__ hipfftComplex sh_samples[];
   
   auto group = cg::this_thread_block();
   int thread_index = group.thread_rank();

   sh_samples[thread_index] = samples[thread_index];

   //if ( group.thread_rank() == 0 ) {
   cookbook_fft64( frequencies, sh_samples, num_samples );
   //}
}
